
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void square(float *d_out,float *d_in){
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f*f;
}
int main(){
    const int size = 64;
    const int bytes = size*sizeof(float);
    float h_in[size];
    for (int i = 0; i<size; i++){
        h_in[i] = float(i);
    }
    float h_out[size];
    float *d_in;
    float *d_out;
    hipMalloc((void **)&d_in,bytes);
    hipMalloc((void **)&d_out,bytes);
    hipMemcpy(d_in,h_in,bytes,hipMemcpyHostToDevice);
    square<<<1 , size>>>(d_out,d_in);
    hipMemcpy(h_out,d_out,bytes,hipMemcpyDeviceToHost);
    for(int i = 0; i<size;i++){
        printf("%f",h_out[i]);
        printf(((i%4) != 3)?"\t":"\n");
    }
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}